#include "hip/hip_runtime.h"
#include <iostream>
#include "common/book.h"
#include "hip/hip_runtime.h"
#include ""
#include "common/cpu_bitmap.h"
#include "common/cpu_anim.h"

using namespace std;

__global__ void add(int a, int b, int* c)
{
    *c = a + b;
}

__global__ void add2(int* const a, int* const b, int* const c, int N) {
    unsigned int tid = blockIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main1() {
    int c;
    int* dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

    add<<<1, 1>>>(2, 7, dev_c);

    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    cout << "2 + 7 = " << c << endl;

    hipFree(dev_c);

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));

    cout << "Has " << count << " device(s)" << endl;

    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;

        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        cout << prop.name << endl;
        cout << prop.totalGlobalMem / 1024.0 / 1024.0 << endl;
        cout << prop.maxThreadsPerBlock << endl;
        cout << "compute capability: " << prop.major << "." << prop.minor << endl;
    }

    return 0;
}

int main2() {
    const int N = 20;
    int a[N], b[N], c[N];
    int* dev_a, * dev_b, * dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; ++i) {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add2<<<N, 1 >>>(dev_a, dev_b, dev_c, N);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

#define DIM 1000

struct hipComplex {
    float r;
    float i;

    __device__ hipComplex(float a, float b) : r(a), i(b) {}

    __device__ float magnitude2() {
        return r * r + i * i;
    }

    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y) {
    const float scale = 1.5;

    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; ++i) {
        a = a * a + c;
        if (a.magnitude2() > 1000) {
            return 0;
        }
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr) {
    int x = blockIdx.x;
    int y = blockIdx.y;

    int offset = x + y * gridDim.x;

    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

int main3() {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>> (dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

    bitmap.display_and_exit();

    HANDLE_ERROR(hipFree(dev_bitmap));

    return 0;
}

int main() {
    return main3();
}